/*
 * Simplified simulation of high-energy particle storms
 *
 * Parallel computing (Degree in Computer Engineering)
 * 2017/2018
 *
 * Version: 2.0
 *
 * Code prepared to be used with the Tablon on-line judge.
 * The current Parallel Computing course includes contests using:
 * OpenMP, MPI, and CUDA.
 *
 * (c) 2018 Arturo Gonzalez-Escribano, Eduardo Rodriguez-Gutiez
 * Grupo Trasgo, Universidad de Valladolid (Spain)
 *
 * This work is licensed under a Creative Commons Attribution-ShareAlike 4.0 International License.
 * https://creativecommons.org/licenses/by-sa/4.0/
 */
#include<stdio.h>
#include<stdlib.h>
#include<math.h>
#include<sys/time.h>

/* Headers for the CUDA assignment versions */
#include<hip/hip_runtime.h>

/* Use fopen function in local tests. The Tablon online judge software 
   substitutes it by a different function to run in its sandbox */
#ifdef CP_TABLON
#include "cputilstablon.h"
#else
#define    cp_open_file(name) fopen(name,"r")
#endif

/* Function to get wall time */
double cp_Wtime(){
    struct timeval tv;
    gettimeofday(&tv, NULL);
    return tv.tv_sec + 1.0e-6 * tv.tv_usec;
}


#define THRESHOLD    0.001f
#define THREADS_PER_BLOCK 128

/* Structure used to store data for one storm of particles */
typedef struct {
    int size;    // Number of particles
    int *posval; // Positions and values
} Storm;


/* ANCILLARY FUNCTIONS: These are not called from the code section which is measured, leave untouched */
/* DEBUG function: Prints the layer status */
void debug_print(int layer_size, float *layer, int *positions, float *maximum, int num_storms ) {
    int i,k;
    /* Only print for array size up to 35 (change it for bigger sizes if needed) */
    if ( layer_size <= 35 ) {
        /* Traverse layer */
        for( k=0; k<layer_size; k++ ) {
            /* Print the energy value of the current cell */
            printf("%10.4f |", layer[k] );

            /* Compute the number of characters. 
               This number is normalized, the maximum level is depicted with 60 characters */
            int ticks = (int)( 60 * layer[k] / maximum[num_storms-1] );

            /* Print all characters except the last one */
            for (i=0; i<ticks-1; i++ ) printf("o");

            /* If the cell is a local maximum print a special trailing character */
            if ( k>0 && k<layer_size-1 && layer[k] > layer[k-1] && layer[k] > layer[k+1] )
                printf("x");
            else
                printf("o");

            /* If the cell is the maximum of any storm, print the storm mark */
            for (i=0; i<num_storms; i++) 
                if ( positions[i] == k ) printf(" M%d", i );

            /* Line feed */
            printf("\n");
        }
    }
}

/*
 * Function: Read data of particle storms from a file
 */
Storm read_storm_file( char *fname ) {
    FILE *fstorm = cp_open_file( fname );
    if ( fstorm == NULL ) {
        fprintf(stderr,"Error: Opening storm file %s\n", fname );
        exit( EXIT_FAILURE );
    }

    Storm storm;    
    int ok = fscanf(fstorm, "%d", &(storm.size) );
    if ( ok != 1 ) {
        fprintf(stderr,"Error: Reading size of storm file %s\n", fname );
        exit( EXIT_FAILURE );
    }

    storm.posval = (int *)malloc( sizeof(int) * storm.size * 2 );
    if ( storm.posval == NULL ) {
        fprintf(stderr,"Error: Allocating memory for storm file %s, with size %d\n", fname, storm.size );
        exit( EXIT_FAILURE );
    }
    
    int elem;
    for ( elem=0; elem<storm.size; elem++ ) {
        ok = fscanf(fstorm, "%d %d\n", 
                    &(storm.posval[elem*2]),
                    &(storm.posval[elem*2+1]) );
        if ( ok != 2 ) {
            fprintf(stderr,"Error: Reading element %d in storm file %s\n", elem, fname );
            exit( EXIT_FAILURE );
        }
    }
    fclose( fstorm );
    
    return storm;
}

 __global__ void initLayer (float* layer, float* layer_copy, int layer_size) { 
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i >= layer_size)
        return;

    layer[i] = 0.0f;
    layer_copy[i] = 0.0f;
}

/* THIS FUNCTION CAN BE MODIFIED */
/* Function to update a single position of the layer */
__global__ void update(float treshhold, float energy, int pos, int layer_size, float *layer) {
    int k = blockIdx.x * blockDim.x + threadIdx.x;

    if(k >= layer_size)
        return;
    /* 1. Compute the absolute value of the distance between the
        impact position and the k-th position of the layer */
    int distance = pos - k;
    if ( distance < 0 ) distance = - distance;

    /* 2. Impact cell has a distance value of 1 */
    distance = distance + 1;

    /* 3. Square root of the distance */
    /* NOTE: Real world atenuation typically depends on the square of the distance.
       We use here a tailored equation that affects a much wider range of cells */
    float atenuacion = sqrtf( (float)distance );

    /* 4. Compute attenuated energy */
    float energy_k = energy / layer_size / atenuacion;

    /* 5. Do not add if its absolute value is lower than the threshold */
    if ( energy_k >= treshhold / layer_size || energy_k <= -treshhold / layer_size )
        layer[k] = layer[k] + energy_k;
}

__global__ void copyLayer (float* layer, float* layer_copy, int layer_size) { 
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i >= layer_size)
        return;
    layer_copy[i] = layer[i];
}

__global__ void updateLayer(int block_size, int radius, float *layer_copy, float *layer){
    __shared__ float temp[THREADS_PER_BLOCK + 2 * 1];
    int gindex = blockIdx.x * blockDim.x + threadIdx.x; 
    int lindex = threadIdx.x + radius; 
    temp[lindex] = layer_copy[gindex]; 
    if (threadIdx.x < radius) { 
        temp[lindex - radius] = layer_copy[gindex - radius]; 
        temp[lindex + block_size] = layer_copy[gindex + block_size]; 
    }
    __syncthreads();
    layer[gindex] = ( temp[lindex-radius] + temp[lindex] + temp[lindex+radius] ) / 3; 
}


//TODO: Optimize 4.3 
__global__ void layer_maximum(int layer_index, int layer_size, float* layer, float* maximum, int* positions){
    
}

/*
 * MAIN PROGRAM
 */
int main(int argc, char *argv[]) {
    int i,j,k;

    /* 1.1. Read arguments */
    if (argc<3) {
        fprintf(stderr,"Usage: %s <size> <storm_1_file> [ <storm_i_file> ] ... \n", argv[0] );
        exit( EXIT_FAILURE );
    }
    
    int layer_size = atoi( argv[1] );
    int num_storms = argc-2;
    Storm storms[ num_storms ];

    /* 1.2. Read storms information */
    for( i=2; i<argc; i++ ) 
        storms[i-2] = read_storm_file( argv[i] );

    /* 1.3. Intialize maximum levels to zero */
    float maximum[ num_storms ];
    int positions[ num_storms ];
    for (i=0; i<num_storms; i++) {
        maximum[i] = 0.0f;
        positions[i] = 0;
    }

    /* 2. Begin time measurement */
    hipSetDevice(0);
    hipDeviceSynchronize();
    double ttotal = cp_Wtime();

    /* START: Do NOT optimize/parallelize the code of the main program above this point */

    /* 3.1 Allocate memory for the layer and initialize to zero */
    float *layer = (float *)malloc( sizeof(float) * layer_size );
    float *layer_copy = (float *)malloc( sizeof(float) * layer_size );
    if ( layer == NULL || layer_copy == NULL ) {
        fprintf(stderr,"Error: Allocating the layer memory\n");
        exit( EXIT_FAILURE );
    }

    /* Allocate memory in the gpu for the layer */
    float *d_layer;
    float *d_layer_copy;
    
    int sizeof_layer = sizeof(float) * layer_size;

    hipMalloc((void **)&d_layer, sizeof_layer);
    hipMalloc((void **)&d_layer_copy, sizeof_layer); 
    
    const auto nb = (layer_size + THREADS_PER_BLOCK - 1)/THREADS_PER_BLOCK;

    /* 3.2 Initialize layer to zero */
    
    hipMemcpy(d_layer, layer, sizeof_layer, hipMemcpyHostToDevice);
    hipMemcpy(d_layer_copy, layer_copy, sizeof_layer, hipMemcpyHostToDevice);

    initLayer<<<nb, THREADS_PER_BLOCK>>>(d_layer, d_layer_copy, layer_size);

    /* 4. Storms simulation */
    for( i=0; i<num_storms; i++) {
        
        /* 4.1. Add impacts energies to layer cells */
        /* For each particle */
        for( j=0; j<storms[i].size; j++ ) {
            /* Get impact energy (expressed in thousandths) */
            float energy = (float)storms[i].posval[j*2+1] * 1000;
            /* Get impact position */
            int position = storms[i].posval[j*2];

            /* For each cell in the layer */
            /* Update the energy value for the cell */
                        
            update<<<nb, THREADS_PER_BLOCK>>>(THRESHOLD, energy, position, layer_size, d_layer);
            
        }
        
        /* 4.2. Energy relaxation between storms */
        /* 4.2.1. Copy values to the ancillary array */

        copyLayer<<<nb, THREADS_PER_BLOCK>>>(d_layer, d_layer_copy, layer_size);

        /* 4.2.2. Update layer using the ancillary values.
                  Skip updating the first and last positions */

        updateLayer<<<nb, THREADS_PER_BLOCK>>>(THREADS_PER_BLOCK, 1, d_layer_copy, d_layer);

        hipMemcpy(layer, d_layer, sizeof_layer, hipMemcpyDeviceToHost);

        /* 4.3. Locate the maximum value in the layer, and its position */
        for( k=1; k<layer_size-1; k++ ) {
            /* Check it only if it is a local maximum */
           if ( layer[k] > layer[k-1] && layer[k] > layer[k+1] ) {
                if ( layer[k] > maximum[i] ) {
                    maximum[i] = layer[k];
                    positions[i] = k;
               }
            }
        }

    }

    hipFree(d_layer);
    hipFree(d_layer_copy);

    /* END: Do NOT optimize/parallelize the code below this point */

    /* 5. End time measurement */
    hipDeviceSynchronize();
    ttotal = cp_Wtime() - ttotal;

    /* 6. DEBUG: Plot the result (only for layers up to 35 points) */
    #ifdef DEBUG
    debug_print( layer_size, layer, positions, maximum, num_storms );
    #endif

    /* 7. Results output, used by the Tablon online judge software */
    printf("\n");
    /* 7.1. Total computation time */
    printf("Time: %lf\n", ttotal );
    /* 7.2. Print the maximum levels */
    printf("Result:");
    for (i=0; i<num_storms; i++)
        printf(" %d %f", positions[i], maximum[i] );
    printf("\n");

    /* 8. Free resources */    
    for( i=0; i<argc-2; i++ )
        free( storms[i].posval );

    /* 9. Program ended successfully */
    return 0;
}
